#include <iostream>
#include <hip/hip_runtime.h>

// Declare a __device__ variable
__device__ int deviceVar[3];

__global__ void kernel()
{
    // Modify the __device__ variable
    deviceVar[0] = 1;
    deviceVar[1] = 2;
    deviceVar[2] = 3;
}

int main()
{
    // Launch the kernel
    kernel<<<1, 1>>>();

    // Copy the __device__ variable to host
    int hostVar[3];
    hipMemcpyFromSymbol(hostVar, HIP_SYMBOL(deviceVar), 3 * sizeof(int), 0, hipMemcpyDeviceToHost);

    // Print the value of the __device__ variable
    std::cout << "Values of deviceVar: " << hostVar[0] << ", " << hostVar[1] << ", " << hostVar[2] << std::endl;

    return 0;
}