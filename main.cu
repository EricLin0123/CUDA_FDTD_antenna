#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#define IE 60
#define JE 100
#define KE 16

const float T = 15e-12;     // 15 picoseconds
const float dt = 0.441e-12; // 0.441 picoseconds
const float t0 = 3 * T;     // 45 picoseconds
const float epsilon = 2.2;  // dielectric constant of the Duroid substrates
const int steps = 1000;

/**
 * @brief Computes the value of a Gaussian function.
 *
 * This function calculates the value of a Gaussian function at a given time `t`,
 * with a specified width `T` and center `t0`.
 *
 * @param t The time at which to evaluate the Gaussian function.
 * @return The value of the Gaussian function at time `t`.
 *
 * The Gaussian function is defined as exp(-((t - t0) / T)^2).
 */
float Gaussian(float t)
{
    return exp(-pow((t - t0) / T, 2));
}

int main()
{
    int i, j, k;
    // D = epsilon E
    // permeability of the dielectric medium is around 1, so skip it.
    float dx[IE][JE][KE], dy[IE][JE][KE], dz[IE][JE][KE];
    float ex[IE][JE][KE], ey[IE][JE][KE], ez[IE][JE][KE];
    float hx[IE][JE][KE], hy[IE][JE][KE], hz[IE][JE][KE];
    dx[i][j][k] = dx[i][j][k] + .5 * (hz[i][j][k] - hz[i][j - 1][k] - hy[i][j][k] + hy[i][j][k - 1]);
    dy[i][j][k] = dz[i][j][k] + .5 * (hx[i][j][k] - hx[i][j][k - 1] - hz[i][j][k] + hz[i - 1][j][k]);
    dz[i][j][k] = dz[i][j][k] + .5 * (hy[i][j][k] - hy[i - 1][j][k] - hx[i][j][k] + hx[i][j - 1][k]);
    hx[i][j][k] = hx[i][j][k] + .5 * (ey[i][j][k + 1] - ey[i][j][k] - ez[i][j + 1][k] + ez[i][j][k]);
    hy[i][j][k] = hy[i][j][k] + .5 * (ez[i + 1][j][k] - ez[i][j][k] - ex[i][j][k + 1] + ex[i][j][k]);
    hz[i][j][k] = hz[i][j][k] + .5 * (ex[i][j + 1][k] - ex[i][j][k] - ey[i + 1][j][k] + ey[i][j][k]);
}