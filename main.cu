/* www.antenna-theory.com */
/* Pete Bevelacqua - EE 517 */

/* This program is a 3D FDTD simulation that will
    model the fields coming off of a microstrip, onto a patch antenna.
    ABC's are 1st order.
*/

// #define STRIP


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

/* dimensions in the X, Y, and Z directions */
#define LIMX 60
#define LIMY 100
#define LIMZ 16

/* time at which source is switched off and ABC turned on */
#define SWITCH1 225 // 405
#define DELAY 0

/* Total number of time steps */
#define totalT 8000

#define PI 3.14159265358979
#define MU0 1.25663706e-6
#define EPS0 8.854e-12
#define EPSR 2.2

/* globally declare fields */
__device__ double Ex[LIMX][LIMY][LIMZ], Ey[LIMX][LIMY][LIMZ], Ez[LIMX][LIMY][LIMZ];
double h_Ez[LIMX][LIMY][LIMZ];
__device__ double Hx[LIMX][LIMY][LIMZ], Hy[LIMX][LIMY][LIMZ], Hz[LIMX][LIMY][LIMZ];

/* globally declare stored field arrays for ABCs */
__device__ double HxABC1[LIMX][LIMZ], HzABC1[LIMX][LIMZ], HyABC2[LIMY][LIMZ], HzABC2[LIMY][LIMZ];
__device__ double HyABC3[LIMY][LIMZ], HzABC3[LIMY][LIMZ], HxABC4[LIMX][LIMZ], HzABC4[LIMX][LIMZ];
__device__ double HxABC5[LIMX][LIMY], HyABC5[LIMX][LIMY], ExABC6[LIMX][LIMZ], EzABC6[LIMX][LIMZ];
__device__ double ExABC5[LIMX][LIMZ], EzABC5[LIMX][LIMZ];

/* Storing the output to calculate S-parameters */
double *d_EzOut;
double *h_EzOut;

/*  I want all variables declared globally */
__device__ int i, j, k, ntime, frame = 0;

/*  Variables defining lattice and time steps, from Sheen, 1990 */
__device__ double delX, delY, delZ, delT;
__device__ double T, T0, temp;
double h_T, h_T0, h_delT;

/*  ABC Coefficients....and the FDTD coefficients */
__device__ double abcFSx, abcFSy, abcFSz, abcDIx, abcDIy, abcDIz, abcBx, abcBy, abcBz, cF, cB, cD;
__device__ double tMUX, tMUY, tMUZ, tEPX, tEPY, tEPZ, tERX, tERY, tERZ, tEBX, tEBY, tEBZ;
int hi, hj, hk, h_ntime, h_frame = 0;
FILE *out;
FILE *outGauss;
/* declaration of functions */
void Initialize();
void UpdateEfields();
void Conductors();
void Source();
void FirstABC();
void UpdateHfields();
void SecondABC();

__global__ void InitializeData(void)
{
    /* Define the Space */
    delX = 0.389e-3;
    delY = 0.400e-3;
    delZ = 0.265e-3;
    delT = 0.441e-12;

    /*  The source parameters */
    T = 15.e-12;

    T0 = 3. * T;

    /* Define Free Space ABC coefficients */
    cF = 1 / sqrt(MU0 * EPS0);
    abcFSx = (delT * cF - delX) / (delT * cF + delX);
    abcFSy = (delT * cF - delY) / (delT * cF + delY);
    abcFSz = (delT * cF - delZ) / (delT * cF + delZ);

    /* Define Dielectric ABC coefficients */
    cD = 1 / sqrt(MU0 * EPS0 * EPSR);
    abcDIx = (delT * cD - delX) / (delT * cD + delX);
    abcDIy = (delT * cD - delY) / (delT * cD + delY);
    abcDIz = (delT * cD - delZ) / (delT * cD + delZ);

    /* Define Boundary ABC coefficients */
    cB = 1 / sqrt(MU0 * EPS0 * (EPSR + 1.) / 2.);
    abcBx = (delT * cB - delX) / (delT * cB + delX);
    abcBy = (delT * cB - delY) / (delT * cB + delY);
    abcBz = (delT * cB - delZ) / (delT * cB + delZ);

    printf("abcBx = %lf, abcBy = %lf, abcBz = %lf\n", abcBx, abcBy, abcBz);

    /* Define H coefficients */
    tMUX = delT / MU0 / delX;
    tMUY = delT / MU0 / delY;
    tMUZ = delT / MU0 / delZ;

    /* E coefficients (Free Space)*/
    tEPX = delT / EPS0 / delX;
    tEPY = delT / EPS0 / delY;
    tEPZ = delT / EPS0 / delZ;

    /* E Coefficients (Dielectric) */
    tERX = delT / EPS0 / EPSR / delX;
    tERY = delT / EPS0 / EPSR / delY;
    tERZ = delT / EPS0 / EPSR / delZ;

    /* E Coefficients (Boundary) */
    tEBX = delT / EPS0 * 2. / (EPSR + 1) / delX;
    tEBY = delT / EPS0 * 2. / (EPSR + 1) / delY;
    tEBZ = delT / EPS0 * 2. / (EPSR + 1) / delZ;
    printf("Pete Rules %lf %lf %lf\n", tEBX, tEBY, tEBZ);
}

__global__ void ntimeplus1()
{
    ntime++;
}
int main()
{
    h_delT = 0.441e-12;
    h_T = 15.e-12;
    h_T0 = 3. * h_T;
    FILE *dump;
    hipError_t err;
    char basename[80] = "junk", filename[100];
#ifdef STRIP
    char outputF[20] = "Incident_strip.txt";
#else
    char outputF[20] = "Incident.txt";
#endif
    char outputGauss[20] = "Gauss.txt";
    outGauss = fopen(outputGauss, "w+");
    out = fopen(outputF, "w+");

    InitializeData<<<1, 1>>>();
    Initialize();
    err = hipMalloc((void **)&d_EzOut, totalT * sizeof(double));
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector d_EzOut (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    h_EzOut = (double *)malloc(totalT * sizeof(double));

    /*Do time stepping */
    hipEvent_t start, stop;
    float milliseconds = 0;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);

    for (h_ntime = 0; h_ntime < totalT; h_ntime++)
    {
        printf("Doing time step %d\r", h_ntime);

        UpdateEfields();
        FirstABC();
        Conductors();
        Source();
        UpdateHfields();
        SecondABC();

        /* Write out E-field */
        hk = 2;
        if (h_ntime % 5 == 0)
        {
            err = hipMemcpyFromSymbol(h_Ez, HIP_SYMBOL(Ez), LIMX * LIMY * LIMZ * sizeof(double), 0, hipMemcpyDeviceToHost);
            if (err != hipSuccess)
            {
                fprintf(stderr, "Failed to copy vector Ez from device to host (error code %s)!\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }
            sprintf(filename, "%s.%d", basename, h_frame++);
            dump = fopen(filename, "w");
            for (hi = 0; hi < LIMX; hi++)
                for (hj = 0; hj < LIMY; hj++)
                {
                    fprintf(dump, "%lf\n", h_Ez[hi][hj][hk]);
                }

            fclose(dump);
        }
        ntimeplus1<<<1, 1>>>();
    } /*End of time stepping*/

    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time for the loop: %f ms\n", milliseconds);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    err = hipMemcpy(h_EzOut, d_EzOut, totalT * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector d_EzOut from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    for (h_ntime = 0; h_ntime < totalT; h_ntime++)
    {
        fprintf(out, "%lf\n", h_EzOut[h_ntime]);
    }

    fclose(out);
    fclose(outGauss);
}

/*  Function:  Initialize Fields   */
/**********************************
 *  Zeros all fields and ABC storage arrays *
 *****************************************/
__global__ void InitializeFields(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < LIMX && j < LIMY && k < LIMZ)
    {
        Ex[i][j][k] = 0.;
        Ey[i][j][k] = 0.;
        Ez[i][j][k] = 0.;
        Hx[i][j][k] = 0.;
        Hy[i][j][k] = 0.;
        Hz[i][j][k] = 0.;
    }
}

__global__ void InitializeABCs(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < LIMX && k < LIMZ)
    {
        if (j == 0)
        {
            HxABC1[i][k] = 0.;
            HzABC1[i][k] = 0.;
            HxABC4[i][k] = 0.;
            HzABC4[i][k] = 0.;
            ExABC6[i][k] = 0.;
            EzABC6[i][k] = 0.;
            ExABC5[i][k] = 0.;
            EzABC5[i][k] = 0.;
        }
    }

    if (j < LIMY && k < LIMZ)
    {
        if (i == 0)
        {
            HyABC2[j][k] = 0.;
            HzABC2[j][k] = 0.;
            HyABC3[j][k] = 0.;
            HzABC3[j][k] = 0.;
        }
    }

    if (i < LIMX && j < LIMY)
    {
        if (k == 0)
        {
            HxABC5[i][j] = 0.;
            HyABC5[i][j] = 0.;
        }
    }
}

void Initialize()
{
    // 2 x 100 x 4 = 800 <= 1024
    dim3 threadsPerBlock3D(2, LIMY, 4);
    // 30 x 1 x 4
    dim3 numBlocks3D((LIMX + threadsPerBlock3D.x - 1) / threadsPerBlock3D.x, (LIMY + threadsPerBlock3D.y - 1) / threadsPerBlock3D.y, (LIMZ + threadsPerBlock3D.z - 1) / threadsPerBlock3D.z);

    InitializeFields<<<numBlocks3D, threadsPerBlock3D>>>();
    InitializeABCs<<<numBlocks3D, threadsPerBlock3D>>>();
}
/*  End Initialize Function **********/

/*  Function:  UpdateEfields()    */
/**********************************
/*  Updates Ex, Ey, and Ez.
 *
 ***********************************/
__global__ void UpdateEx(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < LIMX && 1 <= j && j < LIMY - 1 && 1 <= k && k < LIMZ)
    {
        if (k > 3)
        {
            Ex[i][j][k] += tEPY * (Hz[i][j][k] - Hz[i][j - 1][k]) - tEPZ * (Hy[i][j][k] - Hy[i][j][k - 1]);
        }
        else if (k == 3)
        {
            Ex[i][j][k] += tEBY * (Hz[i][j][k] - Hz[i][j - 1][k]) - tEBZ * (Hy[i][j][k] - Hy[i][j][k - 1]);
        }
        else
        {
            Ex[i][j][k] += tERY * (Hz[i][j][k] - Hz[i][j - 1][k]) - tERZ * (Hy[i][j][k] - Hy[i][j][k - 1]);
        }
    }
}

__global__ void UpdateExSource(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int j = 0;
    if (i < LIMX && 1 <= k && k < LIMZ)
    {
        if (k > 3)
        {
            Ex[i][j][k] += tEPY * 2. * Hz[i][j][k] - tEPZ * (Hy[i][j][k] - Hy[i][j][k - 1]);
        }
        else if (k == 3)
        {
            Ex[i][j][k] += tEBY * 2. * Hz[i][j][k] - tEBZ * (Hy[i][j][k] - Hy[i][j][k - 1]);
        }
        else
        {
            Ex[i][j][k] += tERY * 2. * Hz[i][j][k] - tERZ * (Hy[i][j][k] - Hy[i][j][k - 1]);
        }
    }
}
__global__ void UpdateEy(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (1 <= i && i < LIMX && j < LIMY - 1 && 1 <= k && k < LIMZ)
    {
        if (k > 3)
        {
            Ey[i][j][k] += tEPZ * (Hx[i][j][k] - Hx[i][j][k - 1]) - tEPX * (Hz[i][j][k] - Hz[i - 1][j][k]);
        }
        else if (k == 3)
        {
            Ey[i][j][k] += tEBZ * (Hx[i][j][k] - Hx[i][j][k - 1]) - tEBX * (Hz[i][j][k] - Hz[i - 1][j][k]);
        }
        else
        {
            Ey[i][j][k] += tERZ * (Hx[i][j][k] - Hx[i][j][k - 1]) - tERX * (Hz[i][j][k] - Hz[i - 1][j][k]);
        }
    }
}

__global__ void UpdateEz(double *d_EzOut)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (1 <= i && i < LIMX && 1 <= j && j < LIMY - 1 && k < LIMZ)
    {
        if (k > 2)
        {
            Ez[i][j][k] += tEPX * (Hy[i][j][k] - Hy[i - 1][j][k]) - tEPY * (Hx[i][j][k] - Hx[i][j - 1][k]);
        }
        else if (k < 3)
        {
            Ez[i][j][k] += tERX * (Hy[i][j][k] - Hy[i - 1][j][k]) - tERY * (Hx[i][j][k] - Hx[i][j - 1][k]);
        }
    }
    __syncthreads();
    d_EzOut[ntime] = Ez[22][40][2] + Ez[22][40][1] + Ez[22][40][0];
}

__global__ void UpdateEzSource(double *d_EzOut)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = 0;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    if (1 <= i && i < LIMX && k < LIMZ)
    {
        if (k >= 3)
        {
            Ez[i][j][k] += tEPX * (Hy[i][j][k] - Hy[i - 1][j][k]) - tEPY * 2. * Hx[i][j][k];
        }
        else if (k < 3)
        {
            Ez[i][j][k] += tERX * (Hy[i][j][k] - Hy[i - 1][j][k]) - tERY * 2. * Hx[i][j][k];
        }
    }
    __syncthreads();
    d_EzOut[ntime] = Ez[22][40][2] + Ez[22][40][1] + Ez[22][40][0];
}
void UpdateEfields()
{
    // 2 x 100 x 4 = 800 <= 1024
    dim3 threadsPerBlock3D(2, LIMY, 4);
    // 30 x 1 x 4
    dim3 numBlocks3D((LIMX + threadsPerBlock3D.x - 1) / threadsPerBlock3D.x, (LIMY + threadsPerBlock3D.y - 1) / threadsPerBlock3D.y, (LIMZ + threadsPerBlock3D.z - 1) / threadsPerBlock3D.z);
    // 60 x 16 = 960 <= 1024
    dim3 threadPerBlock2D(LIMX, LIMZ);
    // 1 x 1
    dim3 numBlocks2D((LIMX + threadPerBlock2D.x - 1) / threadPerBlock2D.x, (LIMZ + threadPerBlock2D.y - 1) / threadPerBlock2D.y);
    /* Update Electric Fields */
    UpdateEx<<<numBlocks3D, threadsPerBlock3D>>>();
    if (h_ntime < SWITCH1)
    {
        UpdateExSource<<<numBlocks2D, threadPerBlock2D>>>();
    }
    UpdateEy<<<numBlocks3D, threadsPerBlock3D>>>();
    UpdateEz<<<numBlocks3D, threadsPerBlock3D>>>(d_EzOut);
    if (h_ntime < SWITCH1)
    {
        UpdateEzSource<<<numBlocks2D, threadPerBlock2D>>>(d_EzOut);
    }
}
/* End UpdateEfields function ********************************/

/*  Function:  Conductors ()   */
/******************************
/*  Zeros the tangential (Ex, Ey) fields on the conductor
    surfaces (ground plane, microstrip, antenna)
/**************************************/

__global__ void GroundPlane(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < LIMX && j < LIMY)
    {
        Ex[i][j][0] = 0.;
        Ey[i][j][0] = 0.;
    }
}

__global__ void uStrip(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = 3;
#ifdef STRIP
    if (19 <= i && i <= 25 && j < LIMY)
    {
        if (i < 25)
            Ex[i][j][k] = 0.;
        Ey[i][j][k] = 0.; // i = 25
    }
#else
    if (19 <= i && i <= 25 && j < 50)
    {
        if (i < 25)
            Ex[i][j][k] = 0.;
        Ey[i][j][k] = 0.; // i = 25
    }
#endif
}

__global__ void PatchAntenna(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = 3;
    if (14 <= i && i < 46 && 50 <= j && j < 89)
    {
        Ex[i][j][k] = 0.;
        Ey[i][j][k] = 0.;
    }
    if (i == 46 && 50 <= j && j < 89)
    {
        Ey[i][j][k] = 0.;
    }
    if (j == 89 && 14 <= i && i <= 46)
    {
        Ex[i][j][k] = 0.;
    }
}
void Conductors()
{
    // 10 x 100 = 1000 <= 1024
    dim3 threadPerBlock2D(10, LIMY);
    // 6 x 1
    dim3 numBlocks2D((LIMX + threadPerBlock2D.x - 1) / threadPerBlock2D.x, (LIMY + threadPerBlock2D.y - 1) / threadPerBlock2D.y);
    GroundPlane<<<numBlocks2D, threadPerBlock2D>>>();
    uStrip<<<numBlocks2D, threadPerBlock2D>>>();
#ifndef STRIP
    PatchAntenna<<<numBlocks2D, threadPerBlock2D>>>();
#endif
}

/* End function:  Conductors **************************/

/* Function:  Source ********************************/
/*
/*  Adds in the source *******************************/
__global__ void GaussianSource(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // from 0 to 6 offset by 19
    int k = blockIdx.y * blockDim.y + threadIdx.y; // from 0 to 2 no offset
    int j = 0;

    temp = (ntime * delT - T0) / T;
    Ez[i + 19][j][k] = exp(-temp * temp);
}

__global__ void NoSource(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x; // from 0 to 5 offset by 19
    int k = blockIdx.y * blockDim.y + threadIdx.y; // from 0 to 2 no offset
    int j = 0;

    Ex[i + 19][j][k] = 0.;
}

void Source()
{
    dim3 threadPerBlock(7, 3);
    dim3 threadPerBlock2(6, 3);
    if (h_ntime < SWITCH1)
    {
        GaussianSource<<<1, threadPerBlock>>>(); // 7x3
    }
    else
    {
        NoSource<<<1, threadPerBlock2>>>(); // 6x3
    }

    fprintf(outGauss, "%lf\n", exp(-((h_ntime * h_delT - h_T0) / h_T) * ((h_ntime * h_delT - h_T0) / h_T)));
}

/* End Function:   Source **********************************/

/* Function:  FirstABC() **********************************/
/* ************************************************       */
/* This first ABC is the only one applied to the E-fields.*/
/* Implementation details are in Scheen, 1990.  Performed */
/* after the source is turned off.  Also stores fields    */
/* needed for next round.                                 */

__global__ void ABCY0(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int j = 0;

    if (i < LIMX && k < LIMZ)
    {
        if (k > 3)
        {
            Ex[i][j][k] = ExABC6[i][k] + abcFSy * (Ex[i][j + 1][k] - Ex[i][j][k]);
            Ez[i][j][k] = EzABC6[i][k] + abcFSy * (Ez[i][j + 1][k] - Ez[i][j][k]);
        }
        else if (k == 3)
        {
            Ex[i][j][k] = ExABC6[i][k] + abcBy * (Ex[i][j + 1][k] - Ex[i][j][k]);
            Ez[i][j][k] = EzABC6[i][k] + abcFSy * (Ez[i][j + 1][k] - Ez[i][j][k]);
        }
        else
        {
            Ex[i][j][k] = ExABC6[i][k] + abcDIy * (Ex[i][j + 1][k] - Ex[i][j][k]);
            Ez[i][j][k] = EzABC6[i][k] + abcDIy * (Ez[i][j + 1][k] - Ez[i][j][k]);
        }
    }
}

__global__ void StoreFieldsY0(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int j = 0;

    if (i < LIMX && k < LIMZ)
    {
        ExABC6[i][k] = Ex[i][j + 1][k];
        EzABC6[i][k] = Ez[i][j + 1][k];
    }
}

__global__ void ABCYLIMY(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int j = LIMY - 1;

    if (i < LIMX && k < LIMZ)
    {
        if (k > 3)
        {
            Ex[i][j][k] = ExABC5[i][k] + abcFSy * (Ex[i][j - 1][k] - Ex[i][j][k]);
            Ez[i][j][k] = EzABC5[i][k] + abcFSy * (Ez[i][j - 1][k] - Ez[i][j][k]);
        }
        else if (k == 3)
        {
            Ex[i][j][k] = ExABC5[i][k] + abcBy * (Ex[i][j - 1][k] - Ex[i][j][k]);
            Ez[i][j][k] = EzABC5[i][k] + abcFSy * (Ez[i][j - 1][k] - Ez[i][j][k]);
        }
        else
        {
            Ex[i][j][k] = ExABC5[i][k] + abcDIy * (Ex[i][j - 1][k] - Ex[i][j][k]);
            Ez[i][j][k] = EzABC5[i][k] + abcDIy * (Ez[i][j - 1][k] - Ez[i][j][k]);
        }
    }
}

__global__ void StoreFieldsYLIMY(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int j = LIMY - 1;

    if (i < LIMX && k < LIMZ)
    {
        ExABC5[i][k] = Ex[i][j - 1][k];
        EzABC5[i][k] = Ez[i][j - 1][k];
    }
}

void FirstABC()
{
    // 60 x 16 = 960 <= 1024
    dim3 threadPerBlock2D(LIMX, LIMZ);
    // 1 x 1
    dim3 numBlocks2D((LIMX + threadPerBlock2D.x - 1) / threadPerBlock2D.x, (LIMZ + threadPerBlock2D.y - 1) / threadPerBlock2D.y);
    /* ABC on the wall y=0 */
    if (h_ntime >= SWITCH1 + DELAY)
    {
        ABCY0<<<numBlocks2D, threadPerBlock2D>>>();
    }

    StoreFieldsY0<<<numBlocks2D, threadPerBlock2D>>>();
    ABCYLIMY<<<numBlocks2D, threadPerBlock2D>>>();
    StoreFieldsYLIMY<<<numBlocks2D, threadPerBlock2D>>>();
}
/* End Function:   FirstABC *******************************/

/* Function:  UpdateHfields() *****************************/
/* Updates H-fields.   Nothing special here.  *************/
__global__ void UpdateHx(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < LIMX && j < LIMY - 1 && k < LIMZ - 1)
    {
        Hx[i][j][k] += tMUZ * (Ey[i][j][k + 1] - Ey[i][j][k]) - tMUY * (Ez[i][j + 1][k] - Ez[i][j][k]);
    }
}

__global__ void UpdateHy(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < LIMX - 1 && j < LIMY && k < LIMZ - 1)
    {
        Hy[i][j][k] += tMUX * (Ez[i + 1][j][k] - Ez[i][j][k]) - tMUZ * (Ex[i][j][k + 1] - Ex[i][j][k]);
    }
}

__global__ void UpdateHz(void)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = blockIdx.z * blockDim.z + threadIdx.z;

    if (i < LIMX - 1 && j < LIMY - 1 && k < LIMZ)
    {
        Hz[i][j][k] += tMUY * (Ex[i][j + 1][k] - Ex[i][j][k]) - tMUX * (Ey[i + 1][j][k] - Ey[i][j][k]);
    }
}

void UpdateHfields()
{
    // 2 x 100 x 4 = 800 <= 1024
    dim3 threadsPerBlock3D(2, LIMY, 4);
    // 30 x 1 x 4
    dim3 numBlocks3D((LIMX + threadsPerBlock3D.x - 1) / threadsPerBlock3D.x, (LIMY + threadsPerBlock3D.y - 1) / threadsPerBlock3D.y, (LIMZ + threadsPerBlock3D.z - 1) / threadsPerBlock3D.z);

    UpdateHx<<<numBlocks3D, threadsPerBlock3D>>>();
    UpdateHy<<<numBlocks3D, threadsPerBlock3D>>>();
    UpdateHz<<<numBlocks3D, threadsPerBlock3D>>>();
}
/* End Function:   UpdateHfields() ***********************/

/* Function:  SecondABC() *********************************/
/* Implements the remaining ABCs on the walls X = 0, LIMX */
/* and Y = LIMY, Z = LIMZ.   Also, the required fields are*/
/* then stored.                                           */
__global__ void SecondABC_1()
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int i = 0;

    if (j < LIMY && k < LIMZ)
    {
        if (k > 3)
        {
            Hy[i][j][k] = HyABC2[j][k] + abcFSx * (Hy[i + 1][j][k] - Hy[i][j][k]);
            Hz[i][j][k] = HzABC2[j][k] + abcFSx * (Hz[i + 1][j][k] - Hz[i][j][k]);
        }
        else if (k == 3)
        {
            Hy[i][j][k] = HyABC2[j][k] + abcFSx * (Hy[i + 1][j][k] - Hy[i][j][k]);
            Hz[i][j][k] = HzABC2[j][k] + abcBx * (Hz[i + 1][j][k] - Hz[i][j][k]);
        }
        else
        {
            Hy[i][j][k] = HyABC2[j][k] + abcDIx * (Hy[i + 1][j][k] - Hy[i][j][k]);
            Hz[i][j][k] = HzABC2[j][k] + abcDIx * (Hz[i + 1][j][k] - Hz[i][j][k]);
        }
    }
}

__global__ void SecondABC_2()
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int i = LIMX - 1;

    if (j < LIMY && k < LIMZ)
    {
        if (k > 3)
        {
            Hy[i][j][k] = HyABC3[j][k] + abcFSx * (Hy[i - 1][j][k] - Hy[i][j][k]);
            Hz[i][j][k] = HzABC3[j][k] + abcFSx * (Hz[i - 1][j][k] - Hz[i][j][k]);
        }
        else if (k == 3)
        {
            Hy[i][j][k] = HyABC3[j][k] + abcFSx * (Hy[i - 1][j][k] - Hy[i][j][k]);
            Hz[i][j][k] = HzABC3[j][k] + abcBx * (Hz[i - 1][j][k] - Hz[i][j][k]);
        }
        else
        {
            Hy[i][j][k] = HyABC3[j][k] + abcDIx * (Hy[i - 1][j][k] - Hy[i][j][k]);
            Hz[i][j][k] = HzABC3[j][k] + abcDIx * (Hz[i - 1][j][k] - Hz[i][j][k]);
        }
    }
}

__global__ void SecondABC_3()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = LIMZ - 1;

    if (i < LIMX && j < LIMY)
    {
        if (k > 3)
        {
            Hx[i][j][k] = HxABC5[i][j] + abcFSz * (Hx[i][j][k - 1] - Hx[i][j][k]);
            Hy[i][j][k] = HyABC5[i][j] + abcFSz * (Hy[i][j][k - 1] - Hy[i][j][k]);
        }
        else if (k == 3)
        {
            Hx[i][j][k] = HxABC5[i][j] + abcFSz * (Hx[i][j][k - 1] - Hx[i][j][k]);
            Hy[i][j][k] = HyABC5[i][j] + abcFSz * (Hy[i][j][k - 1] - Hy[i][j][k]);
        }
        else
        {
            Hx[i][j][k] = HxABC5[i][j] + abcDIz * (Hx[i][j][k - 1] - Hx[i][j][k]);
            Hy[i][j][k] = HyABC5[i][j] + abcDIz * (Hy[i][j][k - 1] - Hy[i][j][k]);
        }
    }
}

__global__ void SaveFields_1()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int j = 0;

    if (i < LIMX && k < LIMZ)
    {
        HxABC4[i][k] = Hx[i][j + 1][k];
        HzABC4[i][k] = Hz[i][j + 1][k];
    }
}

__global__ void SaveFields_2()
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int i = 0;

    if (j < LIMY && k < LIMZ)
    {
        HyABC2[j][k] = Hy[i + 1][j][k];
        HzABC2[j][k] = Hz[i + 1][j][k];
    }
}

__global__ void SaveFields_3()
{
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int i = LIMX - 1;

    if (j < LIMY && k < LIMZ)
    {
        HyABC3[j][k] = Hy[i - 1][j][k];
        HzABC3[j][k] = Hz[i - 1][j][k];
    }
}

__global__ void SaveFields_4()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int k = blockIdx.y * blockDim.y + threadIdx.y;
    int j = LIMY - 1;

    if (i < LIMX && k < LIMZ)
    {
        HxABC1[i][k] = Hx[i][j - 1][k];
        HzABC1[i][k] = Hz[i][j - 1][k];
    }
}

__global__ void SaveFields_5()
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    int k = LIMZ - 1;

    if (i < LIMX && j < LIMY)
    {
        HxABC5[i][j] = Hx[i][j][k - 1];
        HyABC5[i][j] = Hy[i][j][k - 1];
    }
}

void SecondABC()
{
    dim3 threadsPerBlock(4, 4);
    dim3 numBlocks1((LIMY + threadsPerBlock.x - 1) / threadsPerBlock.x, (LIMZ + threadsPerBlock.y - 1) / threadsPerBlock.y);
    dim3 numBlocks2((LIMX + threadsPerBlock.x - 1) / threadsPerBlock.x, (LIMY + threadsPerBlock.y - 1) / threadsPerBlock.y);
    dim3 numBlocks3((LIMX + threadsPerBlock.x - 1) / threadsPerBlock.x, (LIMZ + threadsPerBlock.y - 1) / threadsPerBlock.y);

    SecondABC_1<<<numBlocks1, threadsPerBlock>>>();
    SecondABC_2<<<numBlocks1, threadsPerBlock>>>();
    SecondABC_3<<<numBlocks2, threadsPerBlock>>>();

    SaveFields_1<<<numBlocks3, threadsPerBlock>>>(); // y = 0
    SaveFields_2<<<numBlocks1, threadsPerBlock>>>(); // x = 0
    SaveFields_3<<<numBlocks1, threadsPerBlock>>>(); // x = LIMX - 1
    SaveFields_4<<<numBlocks3, threadsPerBlock>>>(); // y = LIMY - 1
    SaveFields_5<<<numBlocks2, threadsPerBlock>>>(); // z = LIMZ - 1 (top)
}
/* End Function:   SecondABC() *****************************/