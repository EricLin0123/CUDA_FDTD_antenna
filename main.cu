#include <hip/hip_runtime.h>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>

#define IE 60
#define JE 100
#define KE 16

const float T = 15e-12;     // 15 picoseconds
const float dt = 0.441e-12; // 0.441 picoseconds
const float t0 = 3 * T;     // 45 picoseconds
const float epsilon = 2.2;  // dielectric constant of the Duroid substrates
const int steps = 1000;

/**
 * @brief Computes the value of a Gaussian function.
 *
 * This function calculates the value of a Gaussian function at a given time `t`,
 * with a specified width `T` and center `t0`.
 *
 * @param t The time at which to evaluate the Gaussian function.
 * @return The value of the Gaussian function at time `t`.
 *
 * The Gaussian function is defined as exp(-((t - t0) / T)^2).
 */
float Gaussian(float t)
{
    return exp(-pow((t - t0) / T, 2));
}

int main()
{
    int i, j, k;
    int t = 0;
    int nsteps = 1;
    int n = 0;
    float pulse = 0.0f;
    // D = epsilon E
    // permeability of the dielectric medium is around 1, so skip it.
    float ex[IE][JE][KE], ey[IE][JE][KE], ez[IE][JE][KE];
    float hx[IE][JE][KE], hy[IE][JE][KE], hz[IE][JE][KE];

    // Initialize the fields to zero
    for (i = 0; i < IE; i++)
    {
        for (j = 0; j < JE; j++)
        {
            for (k = 0; k < KE; k++)
            {
                ex[i][j][k] = 0.0f;
                ey[i][j][k] = 0.0f;
                ez[i][j][k] = 0.0f;
                hx[i][j][k] = 0.0f;
                hy[i][j][k] = 0.0f;
                hz[i][j][k] = 0.0f;
            }
        }
    }

    // source excitation under the microstrip line
    for (int i = 21; i <= 27; i++)
    {
        for (int k = 0; k <= 3; k++)
        {
            ez[i][0][k] = Gaussian(t);
        }
    }
    while (nsteps > 0)
    {
        printf("nsteps --> ");
        scanf("%d", &nsteps);
        printf("%d \n", nsteps);
        for (n = 1; n <= nsteps; n++)
        {
            t += dt;
            // Update electric fields
            for (i = 1; i < IE; i++)
            {
                for (j = 1; j < JE; j++)
                {
                    for (k = 1; k < KE; k++)
                    {
                        ex[i][j][k] = ex[i][j][k] + .5 * (hz[i][j - 1][k] - hz[i][j][k] - hy[i][j][k] + hy[i][j][k - 1]);
                        ey[i][j][k] = ey[i][j][k] + .5 * (hx[i][j][k] - hx[i][j][k - 1] - hz[i][j][k] + hz[i - 1][j][k]);
                        ez[i][j][k] = ez[i][j][k] + .5 * (hy[i][j][k] - hy[i - 1][j][k] - hx[i][j][k] + hx[i][j - 1][k]);
                    }
                }
            }

            pulse = Gaussian(t);
            // Source excitation
            for (i = 21; i <= 27; i++)
            {
                for (k = 0; k <= 3; k++)
                {
                    ez[i][0][k] = pulse;
                }
            }

            // Update magnetic fields
            for (i = 0; i < IE - 1; i++)
            {
                for (j = 0; j < JE - 1; j++)
                {
                    for (k = 0; k < KE - 1; k++)
                    {
                        hx[i][j][k] = hx[i][j][k] + .5 * (ey[i][j][k + 1] - ey[i][j][k] - ez[i][j + 1][k] + ez[i][j][k]);
                        hy[i][j][k] = hy[i][j][k] + .5 * (ez[i + 1][j][k] - ez[i][j][k] - ex[i][j][k + 1] + ex[i][j][k]);
                        hz[i][j][k] = hz[i][j][k] + .5 * (ex[i][j + 1][k] - ex[i][j][k] - ey[i + 1][j][k] + ey[i][j][k]);
                    }
                }
            }
        }
    }
}